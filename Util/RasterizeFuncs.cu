#include "hip/hip_runtime.h"
/**
 *  CPE 2012
 *  -------------------
 *  Program
 *
 *  Last Modified:
 *  @author Nick Feeney
 */

#include "RasterizeFuncs.h"
#include "RasterizeHelpers.h"
#include "../NewMeshParser/utils.h"
#include <vector>
#define CUDASAFECALL( call )  CUDA_SAFE_CALL( call )
#include "cutil.h"
#define CUDAERRORCHECK() {                   \
   hipError_t err = hipGetLastError();        \
   if( hipSuccess != err){ \
      printf("CudaErrorCheck %d\n", err);           \
      exit(1); \
   } }


__global__ void rasterizeCUDA_Dev( int width, int height, int offx, int offy, int num_tri, pixel *data,
      Vertex *vertices, Triangle *triangles, BoundingBox *boundingBoxes, Color *colors,
      float *depth, unsigned int *mutex )
{
   int triIndex = blockIdx.x * gridDim.x + blockIdx.y;
   if( triIndex >= num_tri )
      return;
   pixel pix;
   int width_bb;
   int height_bb;

   __shared__ Triangle triangle;
   __shared__ BoundingBox box;
   //Vertices for this triangle
   __shared__ Vertex a;
   __shared__ Vertex b;
   __shared__ Vertex c;
   //Colors
   __shared__ Color a_c;
   __shared__ Color b_c;
   __shared__ Color c_c;

   //Fill out all triangle shared data;
   if( threadIdx.x == 0)
      triangle = triangles[triIndex];
   else if (threadIdx.x == 1)
      box = boundingBoxes[triIndex];

   __syncthreads();
   //Fill out all Vertex shared data;
   if( threadIdx.x == 0 )
      a = vertices[triangle.a];
   else if( threadIdx.x == 1 )
      b = vertices[triangle.b];
   else if( threadIdx.x == 2 )
      c = vertices[triangle.c];
   else if( threadIdx.x == 3 )
      a_c = colors[triangle.a];
   else if( threadIdx.x == 4)
      b_c = colors[triangle.b];
   else if( threadIdx.x == 5)
      c_c = colors[triangle.c];
   __syncthreads();

   width_bb = box.xr - box.xl;
   height_bb = box.yr - box.yl;
   int pxIdx, i, j;
   int bb_size = width_bb * height_bb;
   int loop = bb_size / TILE_WIDTH;
   if( loop < ((float)bb_size) / ((float)TILE_WIDTH))
      loop++;

   for( int n = 0; n < loop; n++ )
   {
      pxIdx = n * TILE_WIDTH + threadIdx.x;

      if( pxIdx >= bb_size)
         return;

      i = pxIdx / width_bb + box.yl;
      j = pxIdx % width_bb + box.xl;

      if((i < 0 && i >= height) || (j < 0 && j >= width) || (i + offy >= height) || (j + offx >= width))
         continue;

      //These are alot of shared mem accesses but less registers. Could use register might be faster
      float beta = (float)((a.x-c.x)*(i-c.y) - (j-c.x)*(a.y-c.y))
         /(float)((b.x-a.x)*(c.y-a.y) - (c.x-a.x)*(b.y-a.y));
      float gamma = (float)((b.x-a.x)*(i-a.y) - (j-a.x)*(b.y-a.y))/
         (float)((b.x-a.x)*(c.y-a.y) - (c.x-a.x)*(b.y-a.y));
      float alpha;
      if( beta+gamma <= 1.01 && beta >=-0.01 && gamma >= -0.01 )
         alpha = 1- beta -gamma;
      else
         continue;

      float depthTemp = a.z * alpha + b.z * beta + c.z *gamma;
      pix.r = a_c.r*alpha + b_c.r*beta + c_c.r*gamma;
      pix.g = a_c.g*alpha + b_c.g*beta + c_c.g*gamma;
      pix.b = a_c.b*alpha + b_c.b*beta + c_c.b*gamma;
      /*for( int h = 0; h < TILE_WIDTH; h++ )
        {
        if( threadIdx.x == h )
        {
        while( !atomicInc( &(mutex[(i+offy)*width + j + offx]), 1) ) {};
        }
        __threadfence();
        __syncthreads();
        }
       */
      while( !atomicInc( &(mutex[(i+offy)*width +j + offx]), 1) ) {};
      if( depthTemp > depth[(i+offy)*width + j+offx] )
      {
         depth[(i+offy)*width + j + offx ] = depthTemp;
         data[(i+offy)*width + j + offx] = pix;
      }
      atomicDec( &(mutex[(i+offy)*width + j +offx]), 0 );
   }
}
__global__ void initData( pixel *data, float *depth, int width, int height ){
   int i = blockIdx.x * INIT_WIDTH + threadIdx.x;
   int j = blockIdx.y * INIT_WIDTH + threadIdx.y;

   if( i < width && j < height )
   {
      data[j*width + i].r = 0;
      data[j*width + i].g = 1;
      data[j*width + i].b = 0;
      depth[j*width + i] = -100000;
   }
}
__global__ void blurHor( pixel *data, pixel *output, int width, int height )
{
   float weight[5];
   /*weight[0] = 0.225585938;
     weight[1] = 0.193359375;
     weight[2] = 0.120849609;
     weight[3] = 0.053710938;
     weight[4] = 0.016113281;
    */

   weight[0] = 0.2270270270;
   weight[1] = 0.1945945946;
   weight[2] = 0.1216216216;
   weight[3] = 0.0540540541;
   weight[4] = 0.0162162162;

   int i = blockIdx.y * blockDim.y + threadIdx.y;
   int j = blockIdx.x * blockDim.x + threadIdx.x;

   if (j >= width || i >= height)
      return;

   /*   for( int i = 0; i < height; i++ )
        {
        for( int j = 0; j < width; j++ )
        {*/
   int inIdx = i*width + j;
   int outIdx = j*height + i;
   pixel temp, flow = data[inIdx];
   temp.r = flow.r * weight[0];
   temp.g = flow.g * weight[0];
   temp.b = flow.b * weight[0];
   for( int k = 1; k < 5; k++ )
   {
      int posIndex = j +k;
      int negIndex = j - k;
      if( posIndex >= width )
         posIndex = width-1;
      if( negIndex < 0 )
         negIndex = 0;
      posIndex += i *width;
      negIndex += i *width;
      
      flow = data[posIndex];
      temp.r += flow.r * weight[k];
      temp.g += flow.g * weight[k];
      temp.b += flow.b * weight[k];

      flow = data[negIndex];
      temp.r += flow.r * weight[k];
      temp.g += flow.g * weight[k];
      temp.b += flow.b * weight[k];
   }
   if( temp.r > 1 )
      temp.r = 1;
   if( temp.g > 1 )
      temp.g = 1;
   if( temp.b > 1 )
      temp.b = 1;

   output[outIdx] = temp;
   //      }
   //   }
}
__global__ void blurVer( pixel *data, pixel *output, int width, int height )
{
   double weight[5];
   weight[0] = 0.2270270270;
   weight[1] = 0.1945945946;
   weight[2] = 0.1216216216;
   weight[3] = 0.0540540541;
   weight[4] = 0.0162162162;

   int i = blockIdx.y * blockDim.y + threadIdx.y;
   int j = blockIdx.x * blockDim.x + threadIdx.x;

   if (j >= width || i >= height)
      return;
   /*   for( int i = 0; i < height; i++ )
        {
        for( int j = 0; j < width; j++ )
        {*/
   int index = i*width + j;
   output[index].r = data[index].r * weight[0];
   output[index].g = data[index].g * weight[0];
   output[index].b = data[index].b * weight[0];
   for( int k = 1; k < 5; k++ )
   {
      int posIndex = i +k;
      int negIndex = i - k;
      if( posIndex >= height )
         continue;//posIndex = height-1;
      if( negIndex < 0 )
         continue;//negIndex = 0;
      posIndex = posIndex *width + j;
      negIndex = negIndex *width + j;
      output[index].r += data[posIndex].r * weight[k];
      output[index].r += data[negIndex].r * weight[k];

      output[index].g += data[posIndex].g * weight[k];
      output[index].g += data[negIndex].g * weight[k];

      output[index].b += data[posIndex].b * weight[k];
      output[index].b += data[negIndex].b * weight[k];
   }
   if( output[index].r > 1 )
      output[index].r = 1;
   if( output[index].g > 1 )
      output[index].g = 1;
   if( output[index].b > 1 )
      output[index].b = 1;
   //      }
   //   }
}
int rasterize( BasicModel &mesh, Tga &file )
{
   hipFuncSetCacheConfig(reinterpret_cast<const void*>( blurHor), hipFuncCachePreferL1 );
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEvent_t start1, stop1;
   hipEvent_t start2, stop2;
   hipEventCreate(&start1);
   hipEventCreate(&stop1);

   hipEventCreate(&start2);
   hipEventCreate(&stop2);

   Normal light;
   light.x = 3;
   light.y = 3;
   light.z = 3;
   int width = file.getWidth();
   int height = file.getHeight();
   //   pixel *tempBuffer = (pixel *) malloc(sizeof(pixel) * width *height );
   pixel *data = file.getBuffer();
   unsigned int tris = mesh.Triangles.size();

   //Converts mesh verts to screenspace
   Vertex *vertices = convertVertices( mesh, width, height );

   BoundingBox *boundingBoxes;
   Triangle *triangles = createTriangles( mesh, &boundingBoxes, vertices );
   Normal *normals = createNormals( mesh, (int)mesh.Vertices.size() );
   Color *colors = createColors( mesh, normals, light );

   Vertex *d_vert;
   Triangle *d_tri;
   BoundingBox *d_box;
   Color *d_color;
   float *d_depth;
   unsigned int *d_mutex;
   pixel *d_data;
   pixel *d_buff;
   hipEventRecord(start, 0);

   CUDASAFECALL(hipMalloc( (void **)&d_depth, sizeof(float) * width * height ));
   CUDASAFECALL(hipMalloc( (void **)&d_vert, sizeof(Vertex) * mesh.Vertices.size() ));
   CUDASAFECALL(hipMalloc( (void **)&d_tri, sizeof(Triangle) * tris )  );
   CUDASAFECALL(hipMalloc( (void **)&d_box, sizeof(BoundingBox) * tris ));
   CUDASAFECALL(hipMalloc( (void **)&d_data, sizeof(pixel) * width * height ) );
   CUDASAFECALL(hipMalloc( (void **)&d_color, sizeof(Color) * mesh.Vertices.size() ));
   CUDASAFECALL(hipMalloc( (void **)&d_mutex, sizeof(unsigned int) * width * height ));

   int w = width / INIT_WIDTH;
   if( w < (float)width / (float)INIT_WIDTH )
      w++;
   int h = height / INIT_WIDTH;
   if( h < (float)height / (float)INIT_WIDTH )
      h++;
   dim3 dimBlock1( INIT_WIDTH, INIT_WIDTH );
   dim3 dimGrid1( w, h );
   initData<<<dimGrid1, dimBlock1>>>( d_data, d_depth, width, height );

   CUDASAFECALL(hipMemcpyAsync( d_vert, vertices, sizeof(Vertex) * mesh.Vertices.size(), hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpyAsync( d_tri, triangles, sizeof(Triangle) *tris, hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpyAsync( d_box, boundingBoxes, sizeof(BoundingBox) * tris, hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemcpyAsync( d_color, colors, sizeof(Color) * mesh.Vertices.size(), hipMemcpyHostToDevice ));
   CUDASAFECALL(hipMemsetAsync( d_mutex, 0, width * height * sizeof(unsigned int)));

   hipDeviceSynchronize();

   unsigned int x;
   x = sqrt( tris );
   if ( x < sqrt( tris ) )
      x++;
   dim3 dimBlock( TILE_WIDTH );
   dim3 dimGrid( x, x );

   hipEventRecord(start1, 0);
   printf("Starting Kernel\n");
   for( int i = 0; i < 5; i++ )
   {
      for( int j = 0; j < 5; j++ )
      {
         rasterizeCUDA_Dev<<< dimGrid, dimBlock >>>(width, height,width/5 * i, height/5 * j, tris, d_data, d_vert,
               d_tri, d_box, d_color, d_depth, d_mutex );
      }
   }
   CUDAERRORCHECK();

   hipEventRecord(stop1, 0);
   hipEventRecord(start2, 0);

   dim3 dimBlock2(INIT_WIDTH, INIT_WIDTH);
   dim3 dimGrid2((height / INIT_WIDTH) + 1, (width / INIT_WIDTH) + 1);

   CUDA_SAFE_CALL(hipMalloc((void **) &d_buff, sizeof(pixel) * width * height)); 

   for( int i = 0; i < 100; i++ )
   {
      blurHor<<<dimGrid2, dimBlock2>>>( d_data, d_buff, width, height );
      blurHor<<<dimGrid2, dimBlock2>>>( d_buff, d_data, height, width );
   }

   CUDASAFECALL(hipMemcpy( data, d_data, sizeof(pixel) * width * height, hipMemcpyDeviceToHost ));
   hipEventRecord(stop2, 0);

   printf("Ending Kernel\n");
   hipFree( d_vert );
   hipFree( d_tri );
   hipFree( d_box );
   hipFree( d_color );
   hipFree( d_mutex );
   hipFree( d_depth );
   hipFree( d_data );
   hipFree( d_buff );

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);

   float elapsedTime;
   hipEventElapsedTime(&elapsedTime, start, stop);
   printf("Cuda Time: %f\n", elapsedTime);

   hipEventElapsedTime(&elapsedTime, start1, stop1);
   printf("Cuda Time rasterize: %f\n", elapsedTime);

   hipEventElapsedTime(&elapsedTime, start2, stop2);
   printf("Cuda Time Blur: %f\n", elapsedTime);

   hipEventDestroy(start);
   hipEventDestroy(stop);
   hipEventDestroy(start1);
   hipEventDestroy(stop1);

   return 0;
}
